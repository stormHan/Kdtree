#include "hip/hip_runtime.h"
#include "CUDA_KDtree.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <cstdio>

#define CUDA_STACK 100 // fixed size stack elements for each thread, increase as required. Used in SearchAtNodeRange.

void CheckCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}


__device__ float Distance(const Point &a, const Point &b)
{
    float dist = 0;

    for(int i=0; i < KDTREE_DIM; i++) {
        float d = a.coords[i] - b.coords[i];
        dist += d*d;
    }

    return dist;
}

__device__ void SearchAtNode(const CUDA_KDNode *nodes, const int *indexes, const Point *pts, int cur, const Point &query, int *ret_index, float *ret_dist, int *ret_node)
{
    // Finds the first potential candidate

    int best_idx = 0;
    float best_dist = FLT_MAX;

    while(true) {
        int split_axis = nodes[cur].level % KDTREE_DIM;

        if(nodes[cur].left == -1) {
            *ret_node = cur;

            for(int i=0; i < nodes[cur].num_indexes; i++) {
                int idx = indexes[nodes[cur].indexes + i];
                float dist = Distance(query, pts[idx]);
                if(dist < best_dist) {
                    best_dist = dist;
                    best_idx = idx;
                }
            }

            break;
        }
        else if(query.coords[split_axis] < nodes[cur].split_value) {
            cur = nodes[cur].left;
        }
        else {
            cur = nodes[cur].right;
        }
    }

    *ret_index = best_idx;
    *ret_dist = best_dist;
}


__device__ void SearchAtNodeRange(const CUDA_KDNode *nodes, const int *indexes, const Point *pts, const Point &query,int cur, float range, int *ret_index, float *ret_dist)
{
    // Goes through all the nodes that are within "range"

    int best_idx = 0;
    float best_dist = FLT_MAX;

    // Ok, we don't have nice STL vectors to use, and we can't dynamically allocate memory with CUDA??
    // We'll use a fixed length stack, increase this as required
    int to_visit[CUDA_STACK];
    int to_visit_pos = 0;

    to_visit[to_visit_pos++] = cur;

    while(to_visit_pos) {
        int next_search[CUDA_STACK];
        int next_search_pos = 0;

        while(to_visit_pos) {
            cur = to_visit[to_visit_pos-1];
            to_visit_pos--;

            int split_axis = nodes[cur].level % KDTREE_DIM;

            if(nodes[cur].left == -1) {
                for(int i=0; i < nodes[cur].num_indexes; i++) {
                    int idx = indexes[nodes[cur].indexes + i];
                    float d = Distance(query, pts[idx]);

                    if(d < best_dist) {
                        best_dist = d;
                        best_idx = idx;
                    }
                }
            }
            else {
                float d = query.coords[split_axis] - nodes[cur].split_value;

                // There are 3 possible scenarios
                // The hypercircle only intersects the left region
                // The hypercircle only intersects the right region
                // The hypercricle intersects both

                if(fabs(d) > range) {
                    if(d < 0)
                        next_search[next_search_pos++] = nodes[cur].left;
                    else
                        next_search[next_search_pos++] = nodes[cur].right;
                }
                else {
                    next_search[next_search_pos++] = nodes[cur].left;
                    next_search[next_search_pos++] = nodes[cur].right;
                }
            }
        }

        // No memcpy available??
        for(int i=0; i  < next_search_pos; i++)
            to_visit[i] = next_search[i];

        to_visit_pos = next_search_pos;
    }

    *ret_index = best_idx;
    *ret_dist = best_dist;
}


__device__ void Search(const CUDA_KDNode *nodes, const int *indexes, const Point *pts, const Point &query, int *ret_index, float *ret_dist)
{
    // Find the first closest node, this will be the upper bound for the next searches
    int best_node = 0;
    int best_idx = 0;
    float best_dist = FLT_MAX;
    float radius = 0;

    SearchAtNode(nodes, indexes, pts, 0 /* root */, query, &best_idx, &best_dist, &best_node);

    radius = sqrt(best_dist);

    // Now find other possible candidates
    int cur = best_node;

    while(nodes[cur].parent != -1) {
        // Go up
        int parent = nodes[cur].parent;
        int split_axis = nodes[parent].level % KDTREE_DIM;

        // Search the other node
        float tmp_dist = FLT_MAX;
        int tmp_idx;

        if(fabs(nodes[parent].split_value - query.coords[split_axis]) <= radius) {
            // Search opposite node
            if(nodes[parent].left != cur)
                SearchAtNodeRange(nodes, indexes, pts, query, nodes[parent].left, radius, &tmp_idx, &tmp_dist);
            else
                SearchAtNodeRange(nodes, indexes, pts, query, nodes[parent].right, radius, &tmp_idx, &tmp_dist);
        }

        if(tmp_dist < best_dist) {
            best_dist = tmp_dist;
            best_idx = tmp_idx;
        }

        cur = parent;
    }

    *ret_index = best_idx;
    *ret_dist = best_dist;
}


__device__ void SearchAtNode_knn(const CUDA_KDNode *nodes, const int *indexes, const Point *pts, int cur, const Point &query, int *ret_index, float *ret_dist, int *ret_node, int k)
{
	int neighbor_nb = 0;

	//travasel to the nodes
	while (true)
	{
		int split_axis = nodes[cur].level % KDTREE_DIM;

		if (nodes[cur].left == -1){
			//Get to the leaf node
			*ret_node = cur;

			neighbor_nb += nodes[cur].num_indexes;
			while (neighbor_nb < k)
			{
				cur = nodes[cur].parent;
				neighbor_nb += nodes[cur].num_indexes;
			}

			//Now we get enough neighbors in cur node
			int *temp_index = (int*)malloc(sizeof(int) * nodes[cur].num_indexes);
			float *temp_dists = (float*)malloc(sizeof(float) * nodes[cur].num_indexes);

			for (int i = 0; i < nodes[cur].num_indexes; ++i)
			{
				temp_index[i] = indexes[nodes[cur].indexes + i];
				temp_dists[i] = Distance(query, pts[temp_index[i]]);
			}

			int n = nodes[cur].num_indexes;
			//����k��ð�ݵõ�ǰС�ľ���
			int best_idx = 0;
			float best_dist = FLT_MAX;
			for (int i = 0; i < k; ++i)
			{
				for (int j = i; j < n; ++j)
				{
					if (temp_dists[j] < best_dist)
					{
						best_dist = temp_dists[j];
						best_idx = temp_index[j];

						temp_dists[j] = temp_dists[i];
						temp_index[j] = temp_index[i];

						temp_index[i] = best_idx;
						temp_dists[i] = best_dist;
					}
				}

				ret_index[i] = best_idx;
				ret_dist[i] = best_dist;

				best_idx = 0;
				best_dist = FLT_MAX;
			}
		}
		else if (query.coords[split_axis] < nodes[cur].split_value){
			cur = nodes[cur].left;
		}
		else{
			cur = nodes[cur].right;
		}
	}
	
}

__device__ void SearchAtiNodeRange_knn(const CUDA_KDNode *nodes, const int *indexes, const Point *pts, const Point &query, int cur, float range, int *ret_index, float *ret_dist, int k)
{

}

__device__ void Search_knn(const CUDA_KDNode *nodes, const int *indexes, const Point *pts, const Point &query, int *ret_index, float *ret_dist, int k)
{
	// Find the first closest node, this will be the upper bound for the next searches
	int best_node = 0;
	int* k_idx = (int*)malloc(sizeof(int) * k);
	float* k_dist = (float*)malloc(sizeof(float) * k);
	float radius = 0;
	
	SearchAtNode_knn(nodes, indexes, pts, 0 /* root */, query, k_idx, k_dist, &best_node, k);

	radius = sqrt(k_dist[k]);

	//Now find other posiible candidates
	int cur = best_node;

	while (nodes[cur].parent != -1)
	{
		int parent = nodes[cur].parent;
		int split_value = nodes[cur].level % KDTREE_DIM;
	}

	free(k_idx);
	free(k_dist);
}

__global__ void SearchBatch(const CUDA_KDNode *nodes, const int *indexes, const Point *pts, int num_pts, Point *queries, int num_queries, int *ret_index, float *ret_dist)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx >= num_queries)
        return;

    Search(nodes, indexes, pts, queries[idx], &ret_index[idx], &ret_dist[idx]);
}


__global__ void SearchBatch_knn(const CUDA_KDNode *nodes, const int *indexes, const Point *pts, int num_pts, Point *queries, int num_queries, int *ret_index, float *ret_dist, int k)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if (idx >= num_queries)
		return;

	Search_knn(nodes, indexes, pts, queries[idx], &ret_index[idx * k], &ret_dist[idx * k], k);
}

CUDA_KDTree::~CUDA_KDTree()
{
    hipFree(m_gpu_nodes);
    hipFree(m_gpu_indexes);
    hipFree(m_gpu_points);
}

void CUDA_KDTree::CreateKDTree(KDNode *root, int num_nodes, const vector <Point> &data)
{
    // Create the nodes again on the CPU, laid out nicely for the GPU transfer
    // Not exactly memory efficient, since we're creating the entire tree again
    m_num_points = data.size();

    hipMalloc((void**)&m_gpu_nodes, sizeof(CUDA_KDNode)*num_nodes);
    hipMalloc((void**)&m_gpu_indexes, sizeof(int)*m_num_points);
    hipMalloc((void**)&m_gpu_points, sizeof(Point)*m_num_points);

    CheckCUDAError("CreateKDTree");

    vector <CUDA_KDNode> cpu_nodes(num_nodes);
    vector <int> indexes(m_num_points);
    vector <KDNode*> to_visit;

    int cur_pos = 0;

    to_visit.push_back(root);

    while(to_visit.size()) {
        vector <KDNode*> next_search;

        while(to_visit.size()) {
            KDNode *cur = to_visit.back();
            to_visit.pop_back();

            int id = cur->id;

            cpu_nodes[id].level = cur->level;
            cpu_nodes[id].parent = cur->_parent;
            cpu_nodes[id].left = cur->_left;
            cpu_nodes[id].right = cur->_right;
            cpu_nodes[id].split_value = cur->split_value;
            cpu_nodes[id].num_indexes = cur->indexes.size();

            if(cur->indexes.size()) {
                for(unsigned int i=0; i < cur->indexes.size(); i++)
                    indexes[cur_pos+i] = cur->indexes[i];

                cpu_nodes[id].indexes = cur_pos;
                cur_pos += cur->indexes.size();
            }
            else {
                cpu_nodes[id].indexes = -1;
            }

            if(cur->left)
                next_search.push_back(cur->left);

            if(cur->right)
                next_search.push_back(cur->right);
        }

        to_visit = next_search;
    }

    hipMemcpy(m_gpu_nodes, &cpu_nodes[0], sizeof(CUDA_KDNode)*cpu_nodes.size(), hipMemcpyHostToDevice);
    hipMemcpy(m_gpu_indexes, &indexes[0], sizeof(int)*indexes.size(), hipMemcpyHostToDevice);
    hipMemcpy(m_gpu_points, &data[0], sizeof(Point)*data.size(), hipMemcpyHostToDevice);

    CheckCUDAError("CreateKDTree");
}

void CUDA_KDTree::Search(const vector <Point> &queries, vector <int> &indexes, vector <float> &dists)
{
    int threads = 512;
    int blocks = queries.size()/threads + ((queries.size() % threads)?1:0);

    Point *gpu_queries;
    int *gpu_ret_indexes;
    float *gpu_ret_dist;

    indexes.resize(queries.size());
    dists.resize(queries.size());

    hipMalloc((void**)&gpu_queries, sizeof(float)*queries.size()*KDTREE_DIM);
    hipMalloc((void**)&gpu_ret_indexes, sizeof(int)*queries.size()*KDTREE_DIM);
    hipMalloc((void**)&gpu_ret_dist, sizeof(float)*queries.size()*KDTREE_DIM);

    CheckCUDAError("Search");

    hipMemcpy(gpu_queries, &queries[0], sizeof(float)*queries.size()*KDTREE_DIM, hipMemcpyHostToDevice);

    CheckCUDAError("Search");

    printf("CUDA blocks/threads: %d %d\n", blocks, threads);

    SearchBatch<<<blocks, threads>>>(m_gpu_nodes, m_gpu_indexes, m_gpu_points, m_num_points, gpu_queries, queries.size(), gpu_ret_indexes, gpu_ret_dist);
    hipDeviceSynchronize();

    CheckCUDAError("Search");

    hipMemcpy(&indexes[0], gpu_ret_indexes, sizeof(int)*queries.size(), hipMemcpyDeviceToHost);
    hipMemcpy(&dists[0], gpu_ret_dist, sizeof(float)*queries.size(), hipMemcpyDeviceToHost);

    hipFree(gpu_queries);
    hipFree(gpu_ret_indexes);
    hipFree(gpu_ret_dist);
}

void CUDA_KDTree::Search_knn(const vector<Point> &queries, vector<int> &indexes, vector<float> &dists, int k)
{
	int threads = 512;
	int blocks = queries.size() / threads + ((queries.size() % threads) ? 1 : 0);

	Point *gpu_queries;
	int *gpu_ret_indexes;
	float *gpu_ret_dist;

	indexes.resize(queries.size() * k);
	dists.resize(queries.size() * k);

	hipMalloc((void**)&gpu_queries, sizeof(float) * queries.size() * KDTREE_DIM);
	hipMalloc((void**)&gpu_ret_indexes, sizeof(int) * k * queries.size());
	hipMalloc((void**)&gpu_ret_dist, sizeof(float) * k * queries.size());
	CheckCUDAError("Initialize the gpu pointer");
	
	//copy the query data
	hipMemcpy(gpu_queries, &queries[0], sizeof(float) * queries.size() * KDTREE_DIM, hipMemcpyHostToDevice);
	CheckCUDAError("Copy the data");

	printf("Cuda blocks / threads : %d %d", blocks, threads);

	SearchBatch_knn << < blocks, threads >> >(m_gpu_nodes, m_gpu_indexes, m_gpu_points, m_num_points, gpu_queries, queries.size(), gpu_ret_indexes, gpu_ret_dist�� k);
	hipDeviceSynchronize();

	CheckCUDAError("kernel function");

	//Copy back the data from GPU
	hipMemcpy(&indexes[0], gpu_ret_indexes, sizeof(int) * queries.size() * k, hipMemcpyDeviceToHost);
	hipMemcpy(&dists[0], gpu_ret_dist, sizeof(float) * k * queries.size(), hipMemcpyDeviceToHost);
	CheckCUDAError("Copy back the data from GPU");

	hipFree(gpu_queries);
	hipFree(gpu_ret_dist);
	hipFree(m_gpu_indexes);

}